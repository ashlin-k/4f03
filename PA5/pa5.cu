#include "hip/hip_runtime.h"
//File: hello.cu

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "ppmFile.h"

//GPU dimensions
int blockX =  5;
int blockY =  5;
int gridX =  5;
int gridY =  5;
int gridZ =  4;





__global__ void vadd(int *redin, int *greenin, int *bluein,
					int *redout, int *greenout, int *blueout,
					int radius) {
					
	int *pixin, *pixout;
//calculate thread ID
	int myID = blockIdx.x * gridDim.z * gridDim.y * blockDim.x * blockDim.y
				+blockIdx.y * gridDim.z * blockDim.x * blockDim.y
				+blockIdx.z * blockDim.x * blockDim.y
				+threadIdx.x * blockDim.y
				+threadIdx.y;
	
	
//calculate AVG
	//find col and row of px
	
		int pixID = blockIdx.y * gridDim.x * blockIdx.x;
		int colMin = blockIdx.x - radius;
		int colMax = blockIdx.x + radius;
		int rowMin = blockIdx.y - radius;
		int rowMax = blockIdx.y + radius;
		
		if (colMin <= 0)
			colMin = 0;
		if (colMax >= gridDim.x)
			colMax = gridDim.x-1;
		if (rowMin <= 0)
			rowMin = 0;
		if (rowMax >= gridDim.y)
			rowMax = gridDim.y-1;

			
		if (threadIdx.x == 0)
			pixin = redin;
		if (threadIdx.x == 1)
			pixin = greenin;
		if (threadIdx.x == 2)
			pixin = bluein;
		//sum
		for(int c = colMin; c <= colMax; c++){
			for(int r = rowMin; r <= rowMax; r++){
				pixout[pixID] += pixin[r * gridDim.x + c];
			}
		}
		
		//avg
		pixout[pixID] = (pixout[pixID] / (colMax - colMin + 1) * (rowMax - rowMin + 1)) % 255;
		
		//check for error
		if(pixout[pixID] < 0)
			pixout[pixID] = 0;

		if (threadIdx.x == 0)
			redout = pixout;
		if (threadIdx.x == 1)
			greenout = pixout;
		if (threadIdx.x == 2)
			blueout = pixout;
			
}


int main(int argc, char** argv) {
printf("test");
//begin timer
clock_t startTime = clock();

//init variables
	Image *image;     		// original image, only the num of rows needed for each process
	Image *finalImage; 		// the final complete image
	int width;				// width of image
	int height;				// height of image
	char* infile, *outfile;	// input.ppm output.ppm 3 A
	unsigned long radius;	// blur radius
	int *red, *blue, *green,
		*redin, *greenin, *bluein,
		*redout, *greenout, *blueout,
		*pixelsToCompute, *pixelsToComputeGPU;

//assign variables
  infile = argv[2];
  outfile = argv[3];
  radius = (unsigned int)atol(argv[1]);
  
//display arguments
    printf("Infile: %s\n", infile);
    printf("Outfile: %s\n", outfile);
    printf("Radius: %lu\n", radius);

//read input image
    image = ImageRead(infile);
    width = image->width;
    height = image->height;
    printf("Image is %ix%i\n", width, height);

//set up image arrays
    red = new int[width*height];
    green = new int[width*height];
    blue = new int[width*height];
	


//assign values to array	
	for (int row = 0; row < height; row++) {
		for (int col = 0; col < width; col++){
			long pos = row * width + col;			
			red[pos] = ImageGetPixel(image, col, row, 0);
			green[pos] = ImageGetPixel(image, col, row, 1);
			blue[pos] = ImageGetPixel(image, col, row, 2);
		}
	}

//allocate memory for CUDA	
	hipMalloc ((void **) &redin, sizeof (int) * width*height);
	hipMalloc ((void **) &greenin, sizeof (int) * width*height);
	hipMalloc ((void **) &bluein, sizeof (int) * width*height);
	hipMalloc ((void **) &redout, sizeof (int) * width*height);
	hipMalloc ((void **) &greenout, sizeof (int) * width*height);
	hipMalloc ((void **) &blueout, sizeof (int) * width*height);

	
//copy relavent arrays to memory
	hipMemcpy (redin, red, sizeof (int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy (greenin, green, sizeof (int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy (bluein, blue, sizeof (int) * width*height, hipMemcpyHostToDevice);

	
//Start GPU
	dim3 block (3);
	dim3 grid (width, height);
	vadd <<<grid,block>>> (redin, greenin, bluein,
							redout, greenout, blueout,
							radius);
							
//stop GPU
	hipDeviceSynchronize();
	hipGetLastError();

//get data
	hipMemcpy (red, redout, sizeof (int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy (green, greenout, sizeof (int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy (blue, blueout, sizeof (int) * width*height, hipMemcpyDeviceToHost);

//construct new image
	finalImage = ImageCreate(width, height);
	for (int row = 0; row < height; row++) {
		for (int col = 0; col < width; col++){
			long pos = row * width + col;			
			ImageSetPixel(finalImage, col, row, 0, red[pos]);
			ImageSetPixel(finalImage, col, row, 1, green[pos]);
			ImageSetPixel(finalImage, col, row, 2, blue[pos]);
		}
	}
	ImageWrite(finalImage, outfile);
	
//cleanup
	hipFree ((void *) redin);
	hipFree ((void *) greenin);
	hipFree ((void *) bluein);
	hipFree ((void *) redout);
	hipFree ((void *) greenout);
	hipFree ((void *) blueout);
	if (image != NULL)free(image);
	if (finalImage != NULL)free(finalImage);
	delete []red;
	delete []green;
	delete []blue;
	hipDeviceReset ();
	

//end timer
clock_t stopTime = clock();
double time_spent = (double)(stopTime - startTime) / CLOCKS_PER_SEC;
printf("execution time: %f\n",time_spent);
	
return 0;
}




